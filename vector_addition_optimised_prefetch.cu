#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;
/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__ 
void initWith(float num, float *a, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
  
    for(int i = index; i < N; i += stride)
    {
      a[i] = num;
    }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

pair <int,int> getGridConfig()
{

  /*
   * nsys should register performance changes when execution configuration
   * is updated.
   */
   int deviceId;
   hipGetDevice(&deviceId); 
    
   hipDeviceProp_t props; 
   hipGetDeviceProperties(&props, deviceId);
 
   int multiProcessorCount, maxThreadsPerBlock;
   maxThreadsPerBlock = props.maxThreadsPerBlock;   
   multiProcessorCount = props.multiProcessorCount; 
 
   printf("multiProcessorCount: %d maxThreadsPerBlock: %d\n",multiProcessorCount,maxThreadsPerBlock);

   return make_pair(maxThreadsPerBlock, multiProcessorCount);
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);
  
  int size_float = sizeof(float);

  printf("Size of Float: %d\n", size_float);

  int threadsPerBlock;
  int numberOfBlocks;
  
  pair<int,int> gridConfig = getGridConfig();

  threadsPerBlock = gridConfig.first;
  numberOfBlocks = gridConfig.second;
  printf("numberOfBlocks: %d threadsPerBlock: %d\n",numberOfBlocks,threadsPerBlock);

  float *a;
  float *b;
  float *c;

  int deviceId;
  hipGetDevice(&deviceId);                                         // The ID of the currently active GPU device.
  
  hipMemPrefetchAsync(a, size, deviceId);  
  hipMemPrefetchAsync(b, size, deviceId);  
  hipMemPrefetchAsync(c, size, deviceId);  

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  
  hipMemPrefetchAsync(c, size, hipCpuDeviceId);
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
